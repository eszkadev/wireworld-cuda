#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void step()
{
}

extern "C" int CUDA_step(void)
{
    hipError_t err = hipSuccess;

    int numElements = 100;
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    step<<<blocksPerGrid, threadsPerBlock>>>();
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch step kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}

